#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "tree23_array_h.cu"
#include "tree23_array_d.cu"
#include "timerc.h"
#include <iostream>

const int treeSize = 10;
const int insertSize = 4;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
	{
	  fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	  if (abort) exit(code);
	}
}

__global__ void gpuInsert( int *t, int *insert, int start, int end, int insertPerBlock ){

  //__shared__ int startIndices[gridDim.x];
  // __shared__ int endIndices[gridDim.x];

  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int root = getChild0(t, 0);
  insert = insert + (insertPerBlock*bid);
  /*
  if( tid < insertPerBlock ){
	int spot = findSpot_arr_d(t, root, insert[tid]);
	if(t[getKey0(t, spot)] < insert[tid]){
	  //start[tid] = getChild0(t, spot);
	  //end[tid] = getChild1(t, spot);}
	else{
	  //start[tid] = getChild0(t, spot);
	  //end[tid] = getChild1(t, spot);
	}
  }
  */
  

}


__global__ void findSpots_gpu( int *t, int *insert, int *startInd, int *endInd, int insertPerBlock ){

  //__shared__ int startIndices[insertPerBlock];
  //__shared__ int endIndices[insertPerBlock];

  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int root = getChild0(t, 0);
  insert = insert + (insertPerBlock*bid);
  startInd = startInd + (insertPerBlock*bid);
  endInd = endInd + (insertPerBlock*bid);
  
  if( tid < insertPerBlock ){
	int spot = findSpot_arr_d(t, root, insert[tid]);
	if(t[getKey0(t, spot)] < insert[tid]){
	  startInd[tid] = getChild0(t, spot);
	  endInd[tid] = getChild1(t, spot);}
	else{
	  startInd[tid] = getChild0(t, spot);
	  endInd[tid] = getChild1(t, spot);
	}
  }

}


int main(){


    //INSERT SET
  int *insertSet = (int *) malloc(insertSize*sizeof(int));
  for(int i = 0; i < insertSize; i++)
	insertSet[i] = 2*i+1;

  /*
  //CREATE INITAL TREES
  int *tCPU = (int *) malloc(14*(treeSize+insertSize)*sizeof(int));

  createRoot_arr(tCPU);
  for(int i = 1; i < treeSize; i++){
	insert_arr(tCPU, 0, 2*i);
  }

  //TIMING VARIABLES
  float cputime;
  
  //CPU INSERT
  cstart();
  for(int i = 0; i < insertSize; i++)
	insert_arr(tCPU, 0, insertSet[i]);
  cend(&cputime);

  print_tree(tCPU);
  */
 
 
  //GPU SETUP
  // reset global variables
  float gputime;
  offsetTotal_h = 0;
  numNodes = 0;
  int *tGPU = (int *) malloc(14*(treeSize+insertSize)*sizeof(int));
  createRoot_arr(tGPU);
  for(int i = 1; i < treeSize; i++){
	insert_arr(tGPU, 0, 2*i);
  }
  //int start = insert_arr_index(tGPU, 0, insertArray[0]);
  //int end = insert_arr_index(tGPU, 0, insertArray[insertSize-1]);

  int num_blocks = 1;
  int threads_per_block = 1024;

  
  int *d_t23;
  int *d_insert;
  int *startInd_d;
  int *endInd_d;

  hipMalloc( (void**)&d_t23, (treeSize+insertSize)*14*sizeof(int));
  hipMalloc( (void**)&d_insert, insertSize*sizeof(int));
  hipMalloc( (void**)&startInd_d, insertSize*sizeof(int));
  hipMalloc( (void**)&endInd_d, insertSize*sizeof(int));  

  hipMemcpy(  d_t23 , tGPU , (treeSize)*14*sizeof(int) , hipMemcpyHostToDevice );
  hipMemcpy(  d_insert , insertSet , (insertSize)*sizeof(int) , hipMemcpyHostToDevice );


  int insertPerBlock = treeSize/num_blocks;  
  //gpuInsert<<< num_blocks, threads_per_block >>>( d_t23, d_insert, start, end, insertPerBlock );
  findSpots_gpu<<< num_blocks, threads_per_block >>>( d_t23, d_insert, startInd_d, endInd_d, insertPerBlock);
  
  int *startInd_h = (int *) malloc(insertSize*sizeof(int));
  int *endInd_h = (int *) malloc(insertSize*sizeof(int));
  int *t23_out = (int *) malloc((insertSize+treeSize)*14*sizeof(int));
  hipMemcpy( (void*) t23_out , (void*) d_t23, (treeSize+insertSize)*14*sizeof(int) , hipMemcpyDeviceToHost );
  hipMemcpy( (void*) endInd_h , (void*) endInd_d, (insertSize)*sizeof(int) , hipMemcpyDeviceToHost );
  hipMemcpy( (void*) startInd_h , (void*) startInd_d, (insertSize)*sizeof(int) , hipMemcpyDeviceToHost );


  for(int i =0; i < insertSize; i++)
	printf("startInd[%d] = %d, endInd[%d] = %d\n", i, startInd_h[i], i, endInd_h[i]);
  
  //print_tree(tCPU);
  //  print_inorder(tCPU,0);

  //printf("\n-----------------------\n");
  //print_tree(t23_out);
 
  return 0;

}
