#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "tree23_array_h.cu"
#include "tree23_array_d.cu"
#include "timerc.h"
#include <iostream>

const int treeSize = 10;
const int insertSize = 4;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
	{
	  fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
	  if (abort) exit(code);
	}
}

__global__ void gpuInsert( int *t, int *insert, int insertPerBlock ){

  __shared__ int insertNode[64];

  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int root = getChild0(t, 0);
  insert = insert + (insertPerBlock*bid);


  if( tid < insertPerBlock ){
	int spot = findSpot_arr_d(t, root, insert[tid]);
	insertNode[tid] = spot;
  }

  __syncthreads();

  /*
  if(tid == insertPerBlock + 1){
	int s1 = 0;
	int f1 = 1;
	int count = 0;
	while(insertNode[s1] == insertNode[f1]){
	  s1++;
	  f1 += 2;
	  count += 2;
  */
  
  if( tid < insertPerBlock ){
	insert_help_arr_d(t, insertNode[tid], createNode_arr_d(t, offsetTotal_d, insert[tid]), -2);
  }
  
}


__global__ void findSpots_gpu( int *t, int *insert, int *startInd, int *endInd, int insertPerBlock ){

  //__shared__ int startIndices[insertPerBlock];
  //__shared__ int endIndices[insertPerBlock];

  int tid = threadIdx.x;
  int bid = blockIdx.x;
  int root = getChild0(t, 0);
  insert = insert + (insertPerBlock*bid);
  startInd = startInd + (insertPerBlock*bid);

  //  int root = getChild0(t, 0);
  //  insert = insert + (insertPerBlock*bid);
 
  if( tid < insertPerBlock ){
	int spot = findSpot_arr_d(t, root, insert[tid]);
	startInd[tid] = spot;
	if(getKey0(t, spot) > insert[tid]) 
	  startInd[tid] = getChild0(t,spot);
	else if(getKey0(t, spot) < insert[tid])
	  startInd[tid] = getChild1(t,spot);
	else
	  startInd[tid] = getChild2(t,spot);
  }
  /*
  if( tid < insertPerBlock ){
	int spot = findSpot_arr_d(t, 0, insert[tid]);
	printf("tid: %d, val: %d, spot: %d\n", tid, insert[tid], spot);
	startInd[tid] = getChild0(t,spot);
	endInd[tid] = spot;
	if(t[getKey0(t, spot)] < insert[tid]){
	  startInd[tid] = getChild0(t, spot);
	  endInd[tid] = getChild1(t, spot);}
	else if(t[getKey1(t, spot)] < insert[tid]){
	  startInd[tid] = getChild1(t, spot);
	  endInd[tid] = getChild2(t, spot);}
	else{
	  int c = getChild2(t, spot);
	  startInd[tid] = c;
	  endInd[tid] = getChild0(t, c);
	  }*/

}


int main(){


    //INSERT SET
  int *insertSet = (int *) malloc(insertSize*sizeof(int));
  for(int i = 0; i < insertSize; i++)
	insertSet[i] = i+10;

  /*
  //CREATE INITAL TREES
  int *tCPU = (int *) malloc(14*(treeSize+insertSize)*sizeof(int));

  createRoot_arr(tCPU);
  for(int i = 1; i < treeSize; i++){
	insert_arr(tCPU, 0, 2*i);
  }

  //TIMING VARIABLES
  float cputime;
  
  //CPU INSERT
  cstart();
  for(int i = 0; i < insertSize; i++)
	insert_arr(tCPU, 0, insertSet[i]);
  cend(&cputime);

  print_tree(tCPU);
  */
 
 
  //GPU SETUP
  // reset global variables
  float gputime;
  offsetTotal_h = 0;
  numNodes = 0;
  int *tGPU = (int *) malloc(14*(treeSize+insertSize)*sizeof(int));
  createRoot_arr(tGPU);
  for(int i = 0; i < treeSize; i++){
	insert_arr(tGPU, 0, 10*i);
  }
  //int start = insert_arr_index(tGPU, 0, insertArray[0]);
  //int end = insert_arr_index(tGPU, 0, insertArray[insertSize-1]);

  int num_blocks = 1;
  int threads_per_block = 1024;

  
  int *d_t23;
  int *d_insert;
  int *startInd_d;
  int *endInd_d;

  hipMalloc( (void**)&d_t23, (treeSize+insertSize)*14*sizeof(int));
  hipMalloc( (void**)&d_insert, insertSize*sizeof(int));
  hipMalloc( (void**)&startInd_d, insertSize*sizeof(int));
  hipMalloc( (void**)&endInd_d, insertSize*sizeof(int));  

  hipMemcpy(  d_t23 , tGPU , (treeSize)*14*sizeof(int) , hipMemcpyHostToDevice );
  hipMemcpy(  d_insert , insertSet , (insertSize)*sizeof(int) , hipMemcpyHostToDevice );


  int insertPerBlock = treeSize/num_blocks;  
  //gpuInsert<<< num_blocks, threads_per_block >>>( d_t23, d_insert, insertPerBlock );
  findSpots_gpu<<< num_blocks, threads_per_block >>>( d_t23, d_insert, startInd_d, endInd_d, insertPerBlock);
  
  int *startInd_h = (int *) malloc(insertSize*sizeof(int));
  int *endInd_h = (int *) malloc(insertSize*sizeof(int));
  int *t23_out = (int *) malloc((insertSize+treeSize)*14*sizeof(int));
  hipMemcpy( (void*) t23_out , (void*) d_t23, (treeSize+insertSize)*14*sizeof(int) , hipMemcpyDeviceToHost );
  hipMemcpy( (void*) endInd_h , (void*) endInd_d, (insertSize)*sizeof(int) , hipMemcpyDeviceToHost );
  hipMemcpy( (void*) startInd_h , (void*) startInd_d, (insertSize)*sizeof(int) , hipMemcpyDeviceToHost );


  //print_tree(tGPU);

  print_tree(tGPU);
  printf("Insert Set: \n");
  
  for(int i =0; i < insertSize; i++)
	printf("%d, ", insertSet[i]);

  printf("\n");

  
  for(int i =0; i < insertSize; i++)
	printf("startInd[%d] = %d, endInd[%d] = %d\n", i, startInd_h[i], i, endInd_h[i]);
 
  //print_tree(tCPU);
  print_inorder(t23_out,0);

  //printf("\n-----------------------\n");
  //print_tree(t23_out);

  printf("\n");
  return 0;

}
